#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

// Define the s array and K constants
__constant__ uint32_t s[64] = {
    7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22,
    5, 9,  14, 20, 5, 9,  14, 20, 5, 9,  14, 20, 5, 9,  14, 20,
    4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23,
    6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21, 6, 10, 15, 21};

__constant__ uint32_t K[64] = {
    0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee, 0xf57c0faf, 0x4787c62a,
    0xa8304613, 0xfd469501, 0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
    0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821, 0xf61e2562, 0xc040b340,
    0x265e5a51, 0xe9b6c7aa, 0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
    0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed, 0xa9e3e905, 0xfcefa3f8,
    0x676f02d9, 0x8d2a4c8a, 0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
    0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70, 0x289b7ec6, 0xeaa127fa,
    0xd4ef3085, 0x04881d05, 0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
    0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039, 0x655b59c3, 0x8f0ccc92,
    0xffeff47d, 0x85845dd1, 0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
    0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391};

__device__ uint32_t leftrotate(uint32_t x, uint32_t c) {
    return (x << c) | (x >> (32 - c));
}

__global__ void md5_kernel(uint8_t *d_msg, size_t msg_len, uint8_t *d_digest) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx * 64 >= msg_len)
        return;

    uint8_t *msg_chunk = d_msg + idx * 64;

    // Process the message in 512-bit chunks
    uint32_t *chunk = (uint32_t *)msg_chunk;
    uint32_t digest_state[4] = {0x67452301, 0xefcdab89, 0x98badcfe, 0x10325476};

    uint32_t A = digest_state[0];
    uint32_t B = digest_state[1];
    uint32_t C = digest_state[2];
    uint32_t D = digest_state[3];

    for (int j = 0; j < 64; j++) {
        uint32_t F, g;
        if (j < 16) {
            F = (B & C) | ((~B) & D);
            g = j;
        } else if (j < 32) {
            F = (D & B) | ((~D) & C);
            g = (5 * j + 1) % 16;
        } else if (j < 48) {
            F = B ^ C ^ D;
            g = (3 * j + 5) % 16;
        } else {
            F = C ^ (B | (~D));
            g = (7 * j) % 16;
        }

        F = F + A + K[j] + chunk[g];
        A = D;
        D = C;
        C = B;
        B = B + leftrotate(F, s[j]);
    }

    digest_state[0] += A;
    digest_state[1] += B;
    digest_state[2] += C;
    digest_state[3] += D;

    // Write the results to global memory
    for (int i = 0; i < 4; i++) {
        d_digest[idx * 16 + i * 4] = digest_state[i] & 0xFF;
        d_digest[idx * 16 + i * 4 + 1] = (digest_state[i] >> 8) & 0xFF;
        d_digest[idx * 16 + i * 4 + 2] = (digest_state[i] >> 16) & 0xFF;
        d_digest[idx * 16 + i * 4 + 3] = (digest_state[i] >> 24) & 0xFF;
    }
}

void md5(uint8_t *msg, size_t len, uint8_t *digest) {
    // Allocate memory on the GPU
    uint8_t *d_msg, *d_digest;
    size_t padded_len = (len + 1 + 8 + 63) & ~63;
    hipError_t err;

    err = hipMalloc(&d_msg, padded_len);
    if (err != hipSuccess) {
        printf("CUDA malloc d_msg failed: %s\n", hipGetErrorString(err));
        return;
    }
    err = hipMalloc(&d_digest, ((padded_len + 63) / 64) * 16);
    if (err != hipSuccess) {
        printf("CUDA malloc d_digest failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Prepare the message
    uint8_t *padded_msg = (uint8_t *)malloc(padded_len);
    memcpy(padded_msg, msg, len);
    padded_msg[len] = 0x80;
    memset(padded_msg + len + 1, 0, padded_len - len - 1 - 8);
    uint64_t bit_len = len * 8;
    memcpy(padded_msg + padded_len - 8, &bit_len, 8);

    // Copy the message to the GPU
    err = hipMemcpy(d_msg, padded_msg, padded_len, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA memcpy to device failed: %s\n", hipGetErrorString(err));
        return;
    }
    free(padded_msg);

    // Launch the kernel with enough threads to cover the entire message
    int blockSize = 256;
    int numBlocks = (padded_len / 64 + blockSize - 1) / blockSize;
    md5_kernel<<<numBlocks, blockSize>>>(d_msg, padded_len, d_digest);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch failed: %s\n", hipGetErrorString(err));
        return;
    }

    // Copy the digest back to the host
    err = hipMemcpy(digest, d_digest, ((padded_len + 63) / 64) * 16,
                     hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("CUDA memcpy to host failed: %s\n", hipGetErrorString(err));
        return;
    }

    hipFree(d_msg);
    hipFree(d_digest);
}

int main(int argc, char *argv[]) {
    uint8_t *initial_msg;
    size_t initial_len;

    if (argc > 1) {
        initial_msg = (uint8_t *)argv[1];
        initial_len = strlen((char *)initial_msg);
    } else {
        printf("Abort. Please specify a input\n");
        return 1;
    }

    uint8_t digest[16];
    md5(initial_msg, initial_len, digest);

    for (int i = 0; i < 16; i++) {
        printf("%02x", digest[i]);
    }
    printf("\n");
    return 0;
}
